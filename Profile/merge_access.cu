#include <bits/stdc++.h>

#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <ctime>
#include <sys/time.h>

#include <hipblas.h>

void __global__ add1(float *x, float *y, float *z){
    int n = threadIdx.x + blockIdx.x * blockDim.x;
    z[n] = x[n] + y[n];
}

void __global__ add2(float *x, float *y, float *z){
    int n = threadIdx.x + blockIdx.x * blockDim.x + 1;
    z[n] = x[n] + y[n];
}

void __global__ add3(float *x, float *y, float *z){
    int tid_permuted = threadIdx.x ^ 0x1;
    int n = threadIdx.x + blockIdx.x * blockDim.x;
    z[n] = x[n] + y[n];
}


int main(){
    const int N = 32 * 1024 * 1024;
    float *input_x = (float *)malloc(N * sizeof(float));
    float *input_y = (float *)malloc(N * sizeof(float));
    float *d_input_x;
    float *d_input_y;
    hipMalloc((void **)&d_input_x, N * sizeof(float));
    hipMalloc((void **)&d_input_y, N * sizeof(float));
    hipMemcpy(d_input_x, input_x, N *sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input_y, input_y, N *sizeof(float), hipMemcpyHostToDevice);

    float *output = (float *)malloc(N * sizeof(float));
    float *d_output;
    hipMalloc((void **)&d_output, N * sizeof(float));

    dim3 grid(N / 256);
    dim3 block(64);

    for (int i = 0; i < 2; i++){
        add1<<<grid, block>>>(d_input_x, d_input_y, d_output);
        hipDeviceSynchronize();
    }

    hipFree(d_input_x);
    hipFree(d_input_y);
    hipFree(d_output);
    free(input_x);
    free(input_y);
    free(output);
    
    return 0;
}