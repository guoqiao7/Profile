#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>

#include <sys/time.h>

#define THREAD_PER_BLOCK 256

int main(){
    const int N = 32 * 1024 * 1024;
    int block_num = N / THREAD_PER_BLOCK;
    float *in = (float *)malloc(N * sizeof(float));
    float *out = (float *)malloc(block_num * sizeof(float));
    float *res = (float *)malloc(block_num * sizeof(float));

    float *d_in, *d_out;
    hipMalloc(&d_in, N * sizeof(float));
    hipMalloc(&d_out, block_num * sizeof(float));


    free(in);
    free(out);
    free(res);
    hipFree(d_in);
    hipFree(d_out);
}