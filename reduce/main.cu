/*
    naive achivement
    just like a tree, a thread processes neighboring data
*/
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
#include "hip/hip_runtime.h"
#include <sys/time.h>
#include "utils.h"

#define THREAD_PER_BLOCK 256

int main(){
    const int N = 32 * 1024 * 1024;
    int block_num = N / THREAD_PER_BLOCK;
    float *in = (float *)malloc(N * sizeof(float));
    float *out = (float *)malloc(block_num * sizeof(float));
    float *res = (float *)malloc(block_num * sizeof(float));

    float *d_in, *d_out;
    hipMalloc(&d_in, N * sizeof(float));
    hipMalloc(&d_out, block_num * sizeof(float));

    // initial data
    for(int i = 0; i < N; i++){
        in[i] = 1;
    }

    // naive cpu 
    for(int i = 0; i < block_num; i++){
        float temp = 0;
        for(int j = 0; j < THREAD_PER_BLOCK; j++){
            temp += in[i * THREAD_PER_BLOCK + j];  
        }
        res[i] = temp;
    }

    // naive gpu 
    hipMemcpy(d_in, in, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(block_num, 1);
    dim3 block(THREAD_PER_BLOCK, 1);

    // ------------------------------------------------------------------------
    reduce0<<<grid, block>>>(d_in, d_out);

    hipMemcpy(out, d_out, block_num*sizeof(float), hipMemcpyDeviceToHost);

    if(check(out, res, block_num)){
        printf("the result of cpu and gpu is equal\n");
    }

    // -------------------------------------------------------------------------
    reduce1<<<grid, block>>>(d_in, d_out);

    hipMemcpy(out, d_out, block_num*sizeof(float), hipMemcpyDeviceToHost);

    if(check(out, res, block_num)){
        printf("the result of cpu and gpu is equal\n");
    }

    // --------------------------------------------------------------------------
    reduce2<<<grid, block>>>(d_in, d_out);

    hipMemcpy(out, d_out, block_num*sizeof(float), hipMemcpyDeviceToHost);

    if(check(out, res, block_num)){
        printf("the result of cpu and gpu is equal\n");
    }

    // --------------------------------------------------------------------------
    int block_num_3 = N / (2 * THREAD_PER_BLOCK);
    float *out_3 = (float *)malloc(block_num_3 * sizeof(float));
    float *res_3 = (float *)malloc(block_num_3 * sizeof(float));

    float *d_out_3;
    hipMalloc(&d_out_3, block_num_3 * sizeof(float));

    for(int i = 0; i < block_num_3; i++){
        float temp = 0;
        for(int j = 0; j < 2 * THREAD_PER_BLOCK; j++){
            temp += in[i * 2 * THREAD_PER_BLOCK + j];  
        }
        res_3[i] = temp;
    }

    dim3 grid_3(block_num_3, 1);
    dim3 block_3(THREAD_PER_BLOCK, 1);

    reduce3<<<grid_3, block_3>>>(d_in, d_out_3);

    hipMemcpy(out_3, d_out_3, block_num_3*sizeof(float), hipMemcpyDeviceToHost);

    if(check(out_3, res_3, block_num_3)){
        printf("the result of cpu and gpu is equal\n");
    }
    
    // --------------------------------------------------------------------------
    reduce4<<<grid_3, block_3>>>(d_in, d_out_3);

    hipMemcpy(out_3, d_out_3, block_num_3*sizeof(float), hipMemcpyDeviceToHost);

    if(check(out_3, res_3, block_num_3)){
        printf("the result of cpu and gpu is equal\n");
    }

    // --------------------------------------------------------------------------
    reduce5<THREAD_PER_BLOCK><<<grid_3, block_3>>>(d_in, d_out_3);

    hipMemcpy(out_3, d_out_3, block_num_3*sizeof(float), hipMemcpyDeviceToHost);

    if(check(out_3, res_3, block_num_3)){
        printf("the result of cpu and gpu is equal\n");
    }
    
    // --------------------------------------------------------------------------
    const int block_num_6 = 1024;
    const int NUM_PER_BLOCK = N / block_num_6;
    const int NUM_PER_THREAD = NUM_PER_BLOCK / THREAD_PER_BLOCK;

    float *out_6 = (float *)malloc(block_num_6 * sizeof(float));
    float *res_6 = (float *)malloc(block_num_6 * sizeof(float));
    float *d_out_6;
    hipMalloc(&d_out_6, block_num_6 * sizeof(float));

    // cpu
    for (int i = 0; i < block_num_6; i++){
        float cur = 0;
        for (int j = 0; j < NUM_PER_BLOCK; j++){
            if (i * NUM_PER_BLOCK + j < N){
                cur += in[i * NUM_PER_BLOCK + j];
            }
        }
        res_6[i] = cur;
    }

    dim3 grid_6(block_num_6, 1);
    dim3 block_6(THREAD_PER_BLOCK, 1);
    reduce6<THREAD_PER_BLOCK, NUM_PER_THREAD><<<grid_6, block_6>>>(d_in, d_out_6, N);

    hipMemcpy(out_6, d_out_6, block_num_6 * sizeof(float), hipMemcpyDeviceToHost);
    if(check(out_6, res_6, block_num_6)){
        printf("the result of cpu and gpu is equal\n");
    }
    
    // --------------------------------------------------------------------------
    int time = 3;
    for (int i = 0; i < time; i++){
        reduce7<THREAD_PER_BLOCK, NUM_PER_THREAD><<<grid_6, block_6>>>(d_in, d_out_6);
    }

    hipMemcpy(out_6, d_out_6, block_num_6 * sizeof(float), hipMemcpyDeviceToHost);
    if(check(out_6, res_6, block_num_6)){
        printf("the result of cpu and gpu is equal\n");
    }
    // --------------------------------------------------------------------------
    

    free(in);
    free(out);
    free(res);
    free(out_3);
    free(out_6);
    free(res_3);
    free(res_6);
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_3);
    hipFree(d_out_6);
}