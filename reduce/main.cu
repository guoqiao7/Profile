/*
    naive achivement
    just like a tree, a thread processes neighboring data
*/
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
#include "hip/hip_runtime.h"
#include <sys/time.h>
#include "utils.h"

#define THREAD_PER_BLOCK 256

int main(){
    const int N = 32 * 1024 * 1024;
    int block_num = N / THREAD_PER_BLOCK;
    float *in = (float *)malloc(N * sizeof(float));
    float *out = (float *)malloc(block_num * sizeof(float));
    float *res = (float *)malloc(block_num * sizeof(float));

    float *d_in, *d_out;
    hipMalloc(&d_in, N * sizeof(float));
    hipMalloc(&d_out, block_num * sizeof(float));

    // initial data
    for(int i = 0; i < N; i++){
        in[i] = 1;
    }

    // naive cpu 
    for(int i = 0; i < block_num; i++){
        float temp = 0;
        for(int j = 0; j < THREAD_PER_BLOCK; j++){
            temp += in[i * THREAD_PER_BLOCK + j];  
        }
        res[i] = temp;
    }

    // naive gpu 
    hipMemcpy(d_in, in, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(block_num, 1);
    dim3 block(THREAD_PER_BLOCK, 1);

    // ------------------------------------------------------------------------
    reduce0<<<grid, block>>>(d_in, d_out);

    hipMemcpy(out, d_out, block_num*sizeof(float), hipMemcpyDeviceToHost);

    if(check(out, res, block_num)){
        printf("the result of cpu and gpu is equal\n");
    }

    // -------------------------------------------------------------------------
    reduce1<<<grid, block>>>(d_in, d_out);

    hipMemcpy(out, d_out, block_num*sizeof(float), hipMemcpyDeviceToHost);

    if(check(out, res, block_num)){
        printf("the result of cpu and gpu is equal\n");
    }

    // --------------------------------------------------------------------------
    reduce2<<<grid, block>>>(d_in, d_out);

    hipMemcpy(out, d_out, block_num*sizeof(float), hipMemcpyDeviceToHost);

    if(check(out, res, block_num)){
        printf("the result of cpu and gpu is equal\n");
    }

    // --------------------------------------------------------------------------
    int block_num_3 = N / (2 * THREAD_PER_BLOCK);
    float *out_3 = (float *)malloc(block_num_3 * sizeof(float));
    float *res_3 = (float *)malloc(block_num_3 * sizeof(float));

    float *d_out_3;
    hipMalloc(&d_out_3, block_num_3 * sizeof(float));

    for(int i = 0; i < block_num_3; i++){
        float temp = 0;
        for(int j = 0; j < 2 * THREAD_PER_BLOCK; j++){
            temp += in[i * 2 * THREAD_PER_BLOCK + j];  
        }
        res_3[i] = temp;
    }

    dim3 grid_3(block_num_3, 1);
    dim3 block_3(THREAD_PER_BLOCK, 1);

    reduce3<<<grid_3, block_3>>>(d_in, d_out_3);

    hipMemcpy(out_3, d_out_3, block_num_3*sizeof(float), hipMemcpyDeviceToHost);

    if(check(out_3, res_3, block_num_3)){
        printf("the result of cpu and gpu is equal\n");
    }
    
    // --------------------------------------------------------------------------
    reduce4<<<grid_3, block_3>>>(d_in, d_out_3);

    hipMemcpy(out_3, d_out_3, block_num_3*sizeof(float), hipMemcpyDeviceToHost);

    if(check(out_3, res_3, block_num_3)){
        printf("the result of cpu and gpu is equal\n");
    }
    // --------------------------------------------------------------------------
    

    free(in);
    free(out);
    free(res);
    free(out_3);
    free(res_3);
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_3);
}