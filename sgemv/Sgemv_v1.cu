#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hipblas.h>

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])
#define CHECK(call){\
    hipError_t e = call;\
    if (e != hipSuccess){\
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", e);   \
        printf("    Error text: %s\n", hipGetErrorString(e));  \
        exit(1);   \
    }\
}

template <unsigned int WarpSize>
__device__ __forceinline__ float warpReduceSum(float sum){
    if (WarpSize >= 32)sum += __shfl_down_sync(0xffffffff, sum, 16);
    if (WarpSize >= 16)sum += __shfl_down_sync(0xffffffff, sum, 8);
    if (WarpSize >= 8)sum += __shfl_down_sync(0xffffffff, sum, 4);
    if (WarpSize >= 4)sum += __shfl_down_sync(0xffffffff, sum, 2);
    if (WarpSize >= 2)sum += __shfl_down_sync(0xffffffff, sum, 1);
    return sum;
}

void initiaMatrix(float *matrix, int size){
    time_t t;
    srand((unsigned)time(&t));
    // time(NULL)可直接获取当前时间
    // srand((unsigned int)time(NULL));

    // 并行填充
    #pragma omp parallel for
    for (int i = 0; i < size; i++){
        matrix[i] = (float)rand() / (float)RAND_MAX;
    }
}

// N >= 128
__global__ void Sgem_v1(float *__restrict__ A, float *__restrict__ x, float *__restrict__ y,const int M, const int N){
    int bx = blockIdx.x;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    const int warp_size = 32;
    int laneId = tx % warp_size;
    int current_row = blockDim.y * bx + ty;

    if(current_row < M){
        float res = 0;
        int iter = ((N + warp_size - 1) / warp_size) / 4;
        if(iter == 0) iter = 1;
        A = &A[current_row * N];
        #pragma unroll
        for(int i = 0; i < iter; i++){
            int current_col = i * warp_size + laneId;
            float4 current_val = reinterpret_cast<float4 *>(A)[current_col];
            float4 current_x = reinterpret_cast<float4 *>(x)[current_col];
            res += current_val.x * current_x.x;
            res += current_val.y * current_x.y;
            res += current_val.z * current_x.z;
            res += current_val.w * current_x.w;
        }
        res = warpReduceSum<warp_size>(res);
        if (laneId == 0) y[current_row] = res;
    }
}

int main(int argc, char** argv){
    if(argc != 3){
        printf("usage: ./main [M] [N]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t N = atoi(argv[2]);

    size_t bytes_A = sizeof(float) * M * N;
    size_t bytes_x = sizeof(float) * N;
    size_t bytes_y = sizeof(float) * M;

    float *h_A = (float *)malloc(bytes_A);
    float *h_x = (float *)malloc(bytes_x);
    float *h_y = (float *)malloc(bytes_y);
    float *h_y1 = (float *)malloc(bytes_y);

    float *d_A;
    float *d_x;
    float *d_y;
    CHECK(hipMalloc(&d_A, bytes_A));
    CHECK(hipMalloc(&d_x, bytes_x));
    CHECK(hipMalloc(&d_y, bytes_y));
    
    initiaMatrix(h_A, M * N);
    initiaMatrix(h_x, N);

    memset(h_y, 0, bytes_y);
    memset(h_y1, 0, bytes_y);

    int loop = 3;
    CHECK(hipMemcpy(d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_x, h_x, bytes_x, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_y, h_y, bytes_y, hipMemcpyHostToDevice));

    dim3 block(32, 4);
    dim3 grid((M + block.y -1) / block.y);
    for (int i = 0; i < loop; i++){
        Sgem_v1<<<grid, block>>>(d_A, d_x, d_y, M, N);
    }
    
    CHECK(hipMemcpy(h_y, d_y, bytes_y, hipMemcpyDeviceToHost));

    hipblasHandle_t blas_handle;
    hipblasCreate(&blas_handle);
    float alpha = 1.0;
    float beta = 0.0;
    CHECK(hipMemcpy(d_y, h_y1, bytes_y, hipMemcpyHostToDevice));
    for (int i = 0; i < loop; i++){
        hipblasSgemv(blas_handle, HIPBLAS_OP_T,
                    N, M, &alpha,
                    d_A, N, d_x, 1, &beta,
                    d_y, 1);
    }
    CHECK(hipMemcpy(h_y1, d_y, bytes_y, hipMemcpyDeviceToHost));
    hipblasDestroy(blas_handle);

    double eps = 1.e-6;
    bool correct = true;
    for (int i = 0; i < M; i++){
        double abs_err = fabs(h_y[i] - h_y1[i]);
        double dot_length = M;
        double abs_val = fabs(h_y[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if(rel_err > eps){
            printf("Unequal! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                    i, h_y[i], h_y1[i], eps);
            correct = false;
            break;
        }
    }

    printf("%s\n", correct ? "Result is equal" : "Result is FAIL");

    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    free(h_A);
    free(h_x);
    free(h_y);
    free(h_y1);
}