#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREAD_PER_BLOCK 256

/*
    naive achivement
    just like a tree, a thread processes neighboring data
*/
__global__ void reduce0(float *d_in, float *d_out){
    __shared__ float sdata[THREAD_PER_BLOCK];
    
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = d_in[i];
    __syncthreads();

    for(unsigned int s = 1; s < blockDim.x; s *= 2){
        if (tid % (2*s) == 0){
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) d_out[blockIdx.x] = sdata[0];
}


/*
    v0存在warp divergent问题，一个block中的所有线程都会执行同一条指令
    如果存在if-else这样的分支，每个线程都会执行
    也有资料说是因为cuda中取余操作费时过多
    v1尽可能地让所有线程走到同一个分支里面
*/
__global__ void reduce1(float *input, float *output){
    __shared__ float sdata[THREAD_PER_BLOCK];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    sdata[tid] = input[i];
    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2){
        int index = 2 * s * tid;
        if (index < blockDim.x){
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }
    if (tid ==0) output[blockIdx.x] = sdata[0];
}


/*
    v1存在bank冲突，同一个warp中有多个线程需要取同一bank中的数，
    例如，第一次迭代中，warp0中，thread0需要取地址0，1的数，thread16需要取地址32，31的数
    v2修改了stride从128到0
*/
__global__ void reduce2(float *d_in,float *d_out){
    __shared__ float sdata[THREAD_PER_BLOCK];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = d_in[i];
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) d_out[blockIdx.x] = sdata[0];
}


/*
    v2存在idle线程，每次迭代结束后，干活的线程会减少一半
    v3在取数到sharedMem的过程中，加入了一次加法，
    减少了block的数量，每个block处理的数据多了一倍，由256增加到512
*/
__global__ void reduce3(float *d_in,float *d_out){
    __shared__ float sdata[THREAD_PER_BLOCK];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    sdata[tid] = d_in[i] + d_in[i + blockDim.x];
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) d_out[blockIdx.x] = sdata[0];
}


/*
    最后几轮迭代，只有warp0在工作，此时syncthreads可能造成浪费
    将最后一维进行展开，减少同步
*/
__device__ void warpReduce(volatile float* cache, unsigned int tid){
    cache[tid]+=cache[tid+32];
    cache[tid]+=cache[tid+16];
    cache[tid]+=cache[tid+8];
    cache[tid]+=cache[tid+4];
    cache[tid]+=cache[tid+2];
    cache[tid]+=cache[tid+1];
}

__global__ void reduce4(float *d_in,float *d_out){
    __shared__ float sdata[THREAD_PER_BLOCK];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    sdata[tid] = d_in[i] + d_in[i + blockDim.x];
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid < 32) warpReduce(sdata, tid);
    if (tid == 0) d_out[blockIdx.x] = sdata[0];
}

/*
    将for循环完全展开，但现代编译器可能已对此进行优化，提升有限
*/
